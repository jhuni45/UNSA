
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define N 4	
#define T 2


//__global__ void sum_matrix(int**&d_mat_a, int**&dd_mat_a,int**&d_mat_b, int**&dd_mat_b,int**&d_mat_c, int**&dd_mat_c, int n, int m){

__global__ void sum_matrix(int** d_mat_a, int** dd_mat_a,int**d_mat_b, int**dd_mat_b,int**d_mat_c, int**dd_mat_c, int n, int m){

    //sum_matrix<<<grid,blockNum>>>(d_mat_a,dd_mat_a,d_mat_b,dd_mat_b,d_mat_c,dd_mat_c,n,m);

				//int x = threadIdx.x + blockIdx.x*blockDim.x;
				//int y = threadIdx.y + blockIdx.y*blockDim.y;
				d_mat_c[0][0] = -1;
				d_mat_c[0][1] = -2;
				d_mat_c[0][2] = -3;

				dd_mat_c[1][0] = -1;
				dd_mat_c[1][1] = -2;
				dd_mat_c[1][2] = -3;
				
}

void create(int**&mat,int n){
	mat = (int **)malloc(sizeof(int*)*n);	
	int i;
	for(i=0;i<n;i++){
		mat[i] = (int*)malloc(sizeof(int)*n);
	}
}

void create2(int** & mat,int n, int m){
	mat = (int** )malloc(sizeof(int*)*n);	
	mat[0] = (int* )malloc(sizeof(int)*n*m);	
	int i;
	for(i=1;i<=n;i++){
		mat[i] = (*mat+i*m);
	}
}

void fill(int** mat,int n){
	int i,j; 
	for(i=0;i<n;i++){
		for(j=0;j<n;j++)
			mat[i][j] = rand()%10;
	}
}

void fill_zero(int** mat,int n, int value=0){
	int i,j; 
	for(i=0;i<n;i++)
		for(j=0;j<n;j++)
			mat[i][j] = value;
}



void print(int** mat,int n){
	int i,j; 
	for(i=0;i<n;i++){
		for(j=0;j<n;j++)
			printf("%d",mat[i][j]);
		printf("\n");
	}
}

/*
 void create_matrix(int**&mat, int**&h_mat, int**&d_mat, int n, int m){
	int size_n=sizeof(int)*n;
	int size_m=sizeof(int)*m;

	h_mat = (int**)malloc(size_n);

	int i;
	for(i=0;i<n;i++){
		printf(">>>>>\n");
		cudaMalloc((void**)& h_mat[i],size_n);
		cudaMemcpy(h_mat[i],&mat[i][0],size_m,cudaMemcpyHostToDevice);
	}

	cudaMalloc((void*** )& d_mat,size_n);
	cudaMemcpy(d_mat,h_mat,size_n,cudaMemcpyHostToDevice);

}*/






int main(){

	int   n = N;
	int   m = N;

	int** h_mat_a;
		create2(h_mat_a,n,m);
		fill(h_mat_a,n);
		print(h_mat_a,n);
	printf("//////////////////\n");

	int** h_mat_b;
		create2(h_mat_b,n,m);
		fill(h_mat_b,n);
		print(h_mat_b,n);
	printf("//////////////////\n");

	int** h_mat_c;
		create2(h_mat_c,n,m);
		//fill_zero(h_mat_c,n);
		fill(h_mat_c,n);
		print(h_mat_c,n);
	printf("//////////////////\n");


	int ** d_mat_a;	int ** dd_mat_a; 
	int ** d_mat_b;	int ** dd_mat_b; 
	int ** d_mat_c;	int ** dd_mat_c; 
	
	int i;
	 
	///////////////////////

    d_mat_a = (int** )malloc(sizeof(int*)*n);
    hipMalloc((void** )& d_mat_a[0], n*sizeof(int)*m);
    hipMemcpy(d_mat_a[0],h_mat_a[0],n*sizeof(int)*m,hipMemcpyHostToDevice);
    for(i=0;i<n;i++){
    	printf(">wwww>>>>\n");
    	d_mat_a[1]= (*h_mat_a+i*m);
    }
    	
    hipMalloc((void*** )& dd_mat_a,sizeof(int*)*n);
    hipMemcpy(dd_mat_a,h_mat_a,sizeof(int)*n,hipMemcpyHostToDevice);    
 
    ///

    d_mat_b = (int** )malloc(sizeof(int*)*n);
    hipMalloc((void** )& d_mat_b[0], n*sizeof(int)*m);
    hipMemcpy(d_mat_b[0],h_mat_b[0],n*sizeof(int)*m,hipMemcpyHostToDevice);
    for(i=0;i<n;i++){
    	printf(">wwww>>>>\n");
    	d_mat_b[1]= (*h_mat_b+i*m);
    }
    	
    hipMalloc((void*** )& dd_mat_b,sizeof(int*)*n);
    hipMemcpy(dd_mat_b,h_mat_b,sizeof(int)*n,hipMemcpyHostToDevice);    

	///

    d_mat_c = (int** )malloc(sizeof(int*)*n);
    hipMalloc((void** )& d_mat_c[0], n*sizeof(int)*m);
    hipMemcpy(d_mat_c[0],h_mat_c[0], n*sizeof(int)*m,hipMemcpyHostToDevice);
    for(i=0;i<n;i++){
    	printf(">wwww>>>>\n");
    	d_mat_c[1]= (*h_mat_c+i*m);
    }
    	
    hipMalloc((void*** )& dd_mat_c,sizeof(int*)*n);
    hipMemcpy(dd_mat_c,h_mat_c,sizeof(int)*n,hipMemcpyHostToDevice);    
  	
    //////////////////////////

	dim3 grid(ceil(N/T),ceil(N/T),1);
	dim3 blockNum(T,T,1);

	sum_matrix<<<grid,blockNum>>>(d_mat_a,dd_mat_a,d_mat_b,dd_mat_b,d_mat_c,dd_mat_c,n,m);

	hipMemcpy(h_mat_c[0],d_mat_c[0],n*sizeof(int)*m,hipMemcpyDeviceToHost);	
	

	printf("///////CCCCCC///////////\n");
	print(h_mat_c,n);


	return 0;
}